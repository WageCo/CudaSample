#include <assert.h>
#include <cstdint>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

/**
 * @brief GPU每个线程内部执行的函数, 矩阵中每个对应的数加1
 *
 * @param pMatrx 矩阵
 * @param n 矩阵大小
 * @return __global__ void
 */
__global__ void AddSelf(float *pMatrx, uint32_t n) {
  pMatrx[threadIdx.x + threadIdx.y * n] += 1;
}

/**
 * @brief 打印矩阵
 *
 * @param pMatrx 矩阵
 * @param n 矩阵大小
 */
void PrintMatrix(float *pMatrx, uint32_t n) {
  for (auto i = 0; i < n; ++i) {
    for (auto j = 0; j < n; ++j) {
      std::cout << pMatrx[j + i * n] << " ";
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;
}

/**
 * @brief 矩阵自增测试
 *
 * @param n 矩阵大小
 */
void MatrixAddSelfTest(uint32_t n) {
  std::cout << std::endl << __func__ << ":" << std::endl;
  // cpu

  // malloc host memory
  float *pHostMatrix = nullptr;
  pHostMatrix = (float *)malloc(sizeof(float) * n * n);
  assert(pHostMatrix);

  // init host memory
  memset(pHostMatrix, 0, sizeof(float) * n * n);
  for (auto i = 0; i < n; ++i) {
    for (auto j = 0; j < n; ++j) {
      pHostMatrix[j + i * n] = 1.0f;
    }
  }
  PrintMatrix(pHostMatrix, n);

  // gpu

  // malloc device memory
  float *pDeviceMatrix = nullptr;
  hipError_t nCudaErr = hipSuccess;
  nCudaErr = hipMalloc(&pDeviceMatrix, sizeof(float) * n * n);
  assert(nCudaErr == hipSuccess);

  // Copy host data to device memory
  nCudaErr = hipMemcpy(pDeviceMatrix, pHostMatrix, sizeof(float) * n * n,
                        hipMemcpyHostToDevice);
  assert(nCudaErr == hipSuccess);

  // Gpu Execute
  dim3 threadBlocks(1, 1, 1);
  dim3 threads(n, n, 1);
  AddSelf<<<threadBlocks, threads>>>(pDeviceMatrix, n);

  // Copy result to host memory
  nCudaErr = hipMemcpy(pHostMatrix, pDeviceMatrix, sizeof(float) * n * n,
                        hipMemcpyDeviceToHost);
  assert(nCudaErr == hipSuccess);
  // Print
  PrintMatrix(pHostMatrix, n);
  free(pHostMatrix);
  hipFree(pDeviceMatrix);
}

/**
 * @brief 获取当前Gpu一些属性
 *
 */
void GetGpuDeviceInfoTest() {
  std::cout << std::endl << __func__ << ":" << std::endl;

  hipDeviceReset();
  int dev = 0;
  hipDeviceProp_t devProp{};
  hipGetDeviceProperties(&devProp, dev);
  std::cout << "使用GPU device " << dev << ": " << devProp.name << std::endl;
  std::cout << "SM的数量：" << devProp.multiProcessorCount << std::endl;
  std::cout << "每个线程块的共享内存大小："
            << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
  std::cout << "每个线程块的最大线程数：" << devProp.maxThreadsPerBlock
            << std::endl;
  std::cout << "每个EM的最大线程数：" << devProp.maxThreadsPerMultiProcessor
            << std::endl;
  std::cout << "每个SM的最大线程束数："
            << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
}

int main(int argc, char *argv[]) {

  // 测试
  GetGpuDeviceInfoTest();

  MatrixAddSelfTest(6);

  return 0;
}